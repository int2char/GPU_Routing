#include "hip/hip_runtime.h"
/*************************************************
Copyright:UESTC
Author: ZhangQian
Date:2016-08-25
Description:GA-PROA method realization
**************************************************/
#include "hip/hip_runtime.h"
#include ""
#include"Graph.h"
#include "service.h"
#include"taskPath.h"
#include"valuemark.h"
#include"hiprand/hiprand_kernel.h"
#include"iostream"
#include <fstream>
#include"const.h"
#include<math.h>
#include"BFS.h"
#include"GAparrel.h"
#include<time.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include"PathArrange.h"
int checksum(int*array){
	int sum = 0;
	for (int i = 0; i < Task; i++)
		sum += array[i];
	return sum;
}
__device__ int Curand(unsigned int*seed, unsigned int offset, int*array)
{
	unsigned long m = 31721474647;
	int a = array[(offset*(*seed))% 99999];
	unsigned long x = (unsigned long)*seed;
	x = (a*x) % m;
	*seed = (unsigned int)a*x;
	return((int)a);
}
__global__ void RawChormes(int*chormes,int *pathnum,int*hops,unsigned int*seed,int*array,float*rawvalue,int*rawmark,float*demand){
	int taskid=blockIdx.y;
	int popid=blockIdx.x*blockDim.x+threadIdx.x;
	if(popid>=pop||taskid>=Task)
		return;

	int choice=Curand(seed,taskid,array)%(pathnum[taskid]+1)-1;
	int Cid=popid*Task+taskid;
	chormes[Cid]=choice;
	rawvalue[Cid]=demand[taskid]/pow(hops[taskid*10+choice],0.5);
	rawmark[Cid]=taskid;
}
__global__ void Cook(int*chormes,int*pathset,int pathd,float*popmcap,float*demand,int*rawmark){
	int popid=blockIdx.x*blockDim.x+threadIdx.x;
	if(popid>=pop)
		return;
	for(int i=0;i<Task;i++)
	{
		int mi=rawmark[popid*Task+i];
		int flag=0;
		int k=chormes[popid*Task + mi];
		int dim = mi* 10 * pathd + k*pathd;
		int j=0;
		int e;
		while(true){
			e=pathset[dim+j];
			if(e<0)
				break;
			if(popmcap[popid*EDge+e]<demand[mi])
			{
				flag=1;
				chormes[popid*Task + mi]=-1;
				break;
			}
			j++;
		}
		if(flag==0)
		{
			j=0;
			while(true)
			{
				e=pathset[dim+j];
				if(e<0)
					break;
				popmcap[popid*EDge+e]-=demand[mi];
				j++;
			}
		}
	}
}
/*************************************************
Function:Fitor
Description:caculate objective of all the chromosomes,if the chromosome solution is not feasible(overflow the link capacity)
the objective of the chromosome will be set to a max value.
*************************************************/
__global__ void Fitor(int*hops,float*capacity,int*chormes,float*demand,int *pathset,int pathd,int*fits_key,float*fits_value){
	int chonum = blockIdx.x;
	int threadid = threadIdx.x;
	int blockdim = blockDim.x;
	__shared__ float f[PERB];
	//caculate load;
	for (int i = threadid; i <PERB; i += blockdim)
		f[i] = 0;
	__syncthreads();
	//count the link capacity bandwidth usage, store the value in shared array f.
	for (int i = threadid; i < Task; i += blockdim)
	{
		int k = chormes[chonum*Task + i];
		if (k>=0){
			float deman = demand[i];
			int j = 0;
			int e;
			int dim = i* 10 * pathd + k*pathd;
			while (true){
				e = pathset[dim + j];
				if (e < 0)
					break;
				atomicAdd(&f[e], deman);
				j++;
			}
		}
	}
	__syncthreads(); 
	//judge if some link is overflow and caculate the cost of the link,
	//store the cost in shared array f.
	for (int i = threadid; i <PERB; i += blockdim)
	{
		
		float deman = 0;
		if (i < Task)
		{
			int k = chormes[chonum*Task + i];
			deman = (k<0) ?(INFHOPS*demand[i]):(hops[i*10+k]*demand[i]);
		}
		//if load is overflow ,demand+=100*Task*INFHOPS
		f[i] = (f[i]>capacity[i])?(deman+100*Task*INFHOPS):deman;
	}
	__syncthreads();
	/*reduce add the link cost*/
	if (PERB> (blockdim))
	{
		for (int i = threadid+blockdim; i <PERB; i += blockdim)
			f[threadid] += f[i];
	}
	__syncthreads();
	int size = (PERB<blockdim) ?PERB: blockdim;
	for (int s = size; s>1; s = (s + 1) / 2)
	{
		if (threadid<s/2)
			f[threadid] += f[threadid + (s + 1) / 2];
		__syncthreads();
	}
	if (threadid == 0){
		fits_value[chonum] = f[0];
		fits_key[chonum] =chonum;
	}
	
}
/*************************************************
Function:GetParents
Description:randomly choose parents in parallel
*************************************************/
__global__ void GetParents(int* parents, int*randarray, unsigned int *seed){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if (id >= 2 * Beta)
		return;
	if (id < Beta)
		parents[id] = Curand(seed, id, randarray) % ALPHA;
	else
		parents[id] = Curand(seed, id, randarray) % (Beta+ ALPHA);
}
/*************************************************
Function:CudaCross
Description:generate children in parallel
*************************************************/
__global__ void CudaCross(int*children, int*chormes,int*fits_key,int*randarray,unsigned int *seed,int*parents){
	unsigned int blockid = blockIdx.y;
	unsigned int threadid = threadIdx.x + blockIdx.x*blockDim.x;
	if (threadid >= Task)
		return;
	unsigned int position = blockid * 2;
	if (position + 1 >=Beta)
		return;
	int monther = parents[blockid];
	int father = parents[blockid+ Beta];
	int mask = Curand(seed,threadid,randarray) % 2;
	if (mask<1)
	{
		children[position*Task + threadid] = chormes[fits_key[father]*Task + threadid];
		children[(position + 1)*Task + threadid] = chormes[fits_key[monther]*Task + threadid];
	}
	else
	{
		children[position*Task + threadid] = chormes[fits_key[monther]*Task + threadid];
		children[(position + 1)*Task + threadid] = chormes[fits_key[father]*Task + threadid];
	}
	__syncthreads();
}
__global__ void GetMu(int*muinfo,int*children, int*chormes,int*randarray, unsigned int *seed, int*pathnum){
	int id = threadIdx.x + blockIdx.x*blockDim.x;
		if (id >=Gama)
			return;
		int muc = Curand(seed, id, randarray) % pop;
		int mup= Curand(seed, id * 13, randarray) % Task;
		int newv = Curand(seed, id * 71, randarray) % (pathnum[mup] + 1);
		if (newv == pathnum[mup])
			newv = -1;
		muinfo[id * 3] = muc;
		muinfo[id * 3 + 1] = mup;
		muinfo[id * 3 + 2] = newv;
}
__global__ void CudaMutation(int*muinfo,int*children, int*chormes, int*pathnum){
	int conum = blockIdx.y;
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id > Task)
		return;
	if (id == muinfo[3 * conum + 1])
		children[(conum+Beta)*Task + id] = muinfo[3 * conum + 2];
	else
		children[(conum + Beta)*Task + id] =chormes[muinfo[3 * conum] * Task + id];
}
__global__ void Reload(int*chormes,int*children,int*fits_key){
	int conum = blockIdx.y;
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (conum>=(Beta+Gama)||id >=Task)
		return;
	chormes[fits_key[conum + ALPHA]*Task + id] = children[conum*Task + id];

}
void NewGAParrel::cudamalloc(){
			hipMalloc((void**)&dev_chormes, Task*pop*sizeof(int));
			hipMalloc((void**)&dev_demand, Task*sizeof(float));
			hipMalloc((void**)&dev_childs, Task*(Beta+Gama)*sizeof(int));
			hipMalloc((void**)&dev_capacity, G.m*sizeof(float));
			hipMalloc((void**)&dev_pathset, Task*taskd*sizeof(int));
			hipMalloc((void**)(&dev_randarray), sizeof(int)*100000);
			hipMalloc((void**)(&dev_parents), sizeof(int)*Beta*2);
			hipMalloc((void**)(&dev_seed), sizeof(unsigned int));
			hipMalloc((void**)(&dev_pathnum), sizeof(int)*Task);
			hipMalloc((void**)(&dev_muinfo), sizeof(int)*Gama*3);
			hipMalloc((void**)(&dev_fit_key), sizeof(int)*pop);
			hipMalloc((void**)(&dev_fit_value), sizeof(float)*pop);
			hipMalloc((void**)(&dev_hops), sizeof(int)*10*Task);
			hipMalloc((void**)(&dev_rawvalue), sizeof(float)*pop*Task);
			hipMalloc((void**)(&dev_rawmark), sizeof(int)*pop*Task);
			hipMalloc((void**)(&dev_popmcap), sizeof(float)*G.m*pop);
			hipMemcpy(dev_rawmark,rawmark, sizeof(int)*pop*Task, hipMemcpyHostToDevice);
			hipMemcpy(dev_rawvalue,rawvalue, sizeof(float)*pop*Task, hipMemcpyHostToDevice);
			hipMemcpy(dev_popmcap,popmcap, sizeof(float)*pop*G.m, hipMemcpyHostToDevice);
			hipMemcpy(dev_capacity,capacity, G.m*sizeof(float), hipMemcpyHostToDevice);
}

void NewGAParrel::cudapre(){
		hipMemcpy(dev_chormes,chormes, Task*pop*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_demand,demand,Task*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_childs,childs,Task*(Beta+Gama)*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_pathset,pathset,Task*taskd*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_randarray, randarray, sizeof(int)*100000, hipMemcpyHostToDevice);
		hipMemcpy(dev_parents,parents, sizeof(int)*Beta*2, hipMemcpyHostToDevice);
		hipMemcpy(dev_seed,seed, sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(dev_pathnum,pathnum, sizeof(int)*Task, hipMemcpyHostToDevice);
		hipMemcpy(dev_muinfo,muinfo, sizeof(int)*Gama*3, hipMemcpyHostToDevice);
		hipMemcpy(dev_fit_key,fit_key, sizeof(int)*pop, hipMemcpyHostToDevice);
		hipMemcpy(dev_fit_value,fit_value, sizeof(float)*pop, hipMemcpyHostToDevice);
		hipMemcpy(dev_hops,hops, sizeof(int)*10*Task, hipMemcpyHostToDevice);
	}
void NewGAParrel::cudafree(){
		hipFree(dev_chormes);
		hipFree(dev_demand);
		hipFree(dev_childs);
		hipFree(dev_capacity);
		hipFree(dev_pathset);
		hipFree(dev_randarray);
		hipFree(dev_parents);
		hipFree(dev_seed);
		hipFree(dev_pathnum);
		hipFree(dev_muinfo);
		hipFree(dev_fit_key);
		hipFree(dev_fit_value);
		hipFree(dev_hops);
		hipFree(dev_rawmark);
		hipFree(dev_rawvalue);
		hipFree(dev_popmcap);
	}
void NewGAParrel::parrelmake(){
	dim3 blocks_s(pop/512 + 1,Task);
	RawChormes<< <blocks_s,512>> >(dev_chormes,dev_pathnum,dev_hops,dev_seed,dev_randarray,dev_rawvalue,dev_rawmark,dev_demand);
	thrust::device_ptr<float> dev_rv(dev_rawvalue);
	thrust::device_ptr<int> dev_rm(dev_rawmark);
		for(int i=0;i<pop;i++)
			thrust::sort_by_key((dev_rv+i*Task),(dev_rv+(i+1)*Task) ,(dev_rm+i*Task),thrust::greater<float>());
	hipMemcpy(rawmark,dev_rawmark, sizeof(int)*pop*Task, hipMemcpyDeviceToHost);
	Cook<< <pop,512>> >(dev_chormes,dev_pathset,pathd,dev_popmcap,dev_demand,dev_rawmark);
}
void NewGAParrel::process(){

}

/*************************************************
Function:GAsearch
Description:parallel GA search realization
*************************************************/
vector<pair<string,float> > NewGAParrel::GAsearch(){
	cudamalloc();
	cout<<"GA Parallel searching......."<<endl;
	float start=float(1000*clock())/ CLOCKS_PER_SEC;
	/*initialize chromosomes*/
	GoldnessMake();
	/*copy data to GPU memory*/
	cudapre();
	best =Task*100*INFHOPS;/*best objective record*/
	int count = 0;
	int iter=0;
	thrust::device_ptr<float> dev_fv(dev_fit_value);
	thrust::device_ptr<int> dev_fk(dev_fit_key);
	vector<float>middata;
	int mkd=2;
	for (int i = 0; i <10000000; i++)
	{
		iter++;
		seed++;
		Fitor << <pop,1024 >> >(dev_hops,dev_capacity, dev_chormes, dev_demand, dev_pathset, pathd, dev_fit_key,dev_fit_value);
		/*sort the chromosomes by objective value*/
		thrust::sort_by_key(dev_fv,dev_fv+pop ,dev_fk,thrust::less<float>());
		hipMemcpy(fit_value,dev_fit_value, sizeof(float)*pop, hipMemcpyDeviceToHost);
		float ans=fit_value[0];
		/*randomly choose parents*/
		GetParents << <(Beta * 2 + 511) / 512, 512 >> >(dev_parents,dev_randarray,dev_seed);
		dim3 blocks_s(Task / 1024 + 1, Beta / 2 + 1);
		/*cross parents chromosomes to get children*/
		CudaCross << <blocks_s, 1024 >> >(dev_childs,dev_chormes,dev_fit_key,dev_randarray,dev_seed,dev_parents);
		GetMu << <Gama / 1024 + 1, 1024 >> >(dev_muinfo,dev_childs,dev_chormes,dev_randarray,dev_seed,dev_pathnum);
		dim3 blocks_s2(Task / 1024 + 1, Gama);
		/*randomly mutate chromosomes*/
		CudaMutation << <blocks_s2, 1024 >> >(dev_muinfo, dev_childs, dev_chormes, dev_pathnum);
		dim3 blocks_s3(Task / 1024 + 1, Gama + Beta);
		Reload << <blocks_s3, 1024 >> >(dev_chormes, dev_childs, dev_fit_key);
		if (ans<best)
		{
			mkd--;
			best = ans;
			count = 0;
		}
		else
			count++;
		middata.push_back(ans);
		if(mkd>0&&count<100)
			continue;
		time_t now=1000*clock()/ CLOCKS_PER_SEC;
		if (count>loomore||((now-start)>EXPIRE&&GANOEX<0))
			break;
	}
	hipMemcpy(chormes,dev_chormes, Task*pop*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(fit_key,dev_fit_key, sizeof(int)*pop, hipMemcpyDeviceToHost);
	float end = float(1000*clock())/ CLOCKS_PER_SEC;
	pair<float,int>md=more();
	vector<pair<string,float>>rdata;
	float lowbound=0;
	for(int i=0;i<Task;i++)
		lowbound+=demand[i]*INFHOPS;
	float gap=middata[middata.size()-1]-best;
	cout<<"gap is"<<gap<<endl;
	for(int i=0;i<middata.size();i++)
		middata[i]-=gap;
	CheckR(&G,Result,serv,string("GA_Paralle"));
	writejsoniter(GAPFILE,middata,string("GA_Paralle"));
	rdata.push_back(make_pair(string("object"),best));
	rdata.push_back(make_pair(string("inf_obj"),lowbound));
	rdata.push_back(make_pair(string("task_add_in"),md.second));
	rdata.push_back(make_pair(string("flow_add_in"),md.first));
	rdata.push_back(make_pair(string("total_weight"),totalweight));
	rdata.push_back(make_pair(string("time"),(end-start)+affier));
	rdata.push_back(make_pair(string("iter_num"),iter));
	rdata.push_back(make_pair(string("iter_time"),float(end-start+affier)/iter));
	rdata.push_back(make_pair(string("gap"),gap));
	writejsondata(DATAFILE,rdata,string("GA_Paralle"));
	return rdata;
}



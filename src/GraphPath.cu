#include "hip/hip_runtime.h"
/*************************************************
Copyright:UESTC
Author: ZhangQian
Date:2016-08-25
Description:realize LR-PROA method
**************************************************/
#include "GraphPath.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include<vector>
#include<algorithm>
#include <utility>
#include <time.h>
#include<math.h>
#include"service.h"
#include"dijkstra.h"
#include"BFS.h"
#include"taskPath.h"
#include"const.h"
#include"routemask.h"
#include"PathArrange.h"
#include<fstream>
using namespace std;
#define threadsize 256/*GPU block thread size*/

bool UDgreater(pair<int, float> elem1, pair<int, float> elem2)
{
	return elem1.second > elem2.second;
}
bool UPGservice(service s1, service s2)
{
	return s1.d>s2.d;
}
bool cmp(float a, float b)
{
	return a<b;
}
/*************************************************
Function:bellmanHigh
Description:Bellman Ford algorithm
edge:edge struct of the graph.
d:distance array of all the services
m:mark if the algorithm has converged or not;if the algorithm has converged,no distance will change.
lambda:duality penalty of the link weight.
mask:store the services ID need to be routed.
stillS:number of services need to be routed.
*************************************************/
__global__ void bellmanHigh(Edge *edge, int *m, float *c, int*p, float*lambda, int*mask, int stillS)
{
	int tid = blockIdx.y;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >= stillS)return;
	i = mask[i];
	int head = edge[tid].head;
	int tail = edge[tid].tail;
	int biao = head*NODE + i;
	float val = c[tail*NODE + i]+1 +lambda[tid];
	if (c[biao] >val){
		*m = 1;
		c[biao] = val;
	}
}
/*************************************************
Function:color
Description:update the precursor node of all nodes.
edge:edge struct of the graph.
p:precursor node array,store the precursor nodes.
d:distance array of all the services
m:mark if the algorithm has converged or not;if the algorithm has converged,no distance will change.
lambda:duality penalty on the weight of link.
mask:store the services ID need to be routed.
stillS:number of services need to be routed.
*************************************************/
__global__ void color(Edge *edge, int *m, float *c, int*p, float*lambda, int *mask, int stillS){

	int tid = blockIdx.y;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >= stillS)return;
	i = mask[i];
	int head = edge[tid].head;
	int tail = edge[tid].tail;
	int biao = head*NODE + i;
	float val = c[tail*NODE + i]+1+lambda[tid];
	if (c[biao] == val){
		p[biao] = tid;
	}
}
/*************************************************
Function:ChangePameterC
Description:initialize the precursor node array and distance array.
d:distance array.
st:start node of services
*************************************************/
__global__ void ChangePameterC(int*p, float*d, int* st){
	int tid = blockIdx.y;
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i >= NODE || tid >= NODE)return;
	int biao = tid*NODE + i;
	d[biao] = (i == tid) ? 0.0 : 10000000000.0;
	p[biao] = -1;
}
/*************************************************
Function:Copy2GPU
Description:initialize and copy data to GPU memory.
*************************************************/
void GraphPath::Copy2GPU(std::vector<service> &s){
	for (int i = 0; i < Task; i++)
	{
		st[i] = s[i].s;
		te[i] = s[i].t;
		pd[i] = (float)s[i].d;
	}
	vector<int>vecmask;
	for (int i = 0; i < Task; i++)
		vecmask.push_back(st[i]);
	vector<int>::iterator begin=vecmask.begin();
	vector<int>::iterator end=unique(vecmask.begin(),vecmask.end());
	stillS=0;
	for(begin;begin<end;begin++)
		mask[stillS++]=*begin;
	for (int i = 0; i < EDge; i++)
		lambda[i] = 0;
	hipMemcpy(dev_st, st, Task*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_te, te, Task*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_lambda, lambda, EDge*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mask, mask, Task*sizeof(int), hipMemcpyHostToDevice);
}
/*************************************************
Function:GraphPath construct function
Description:malloc data memory on GPU and copy data from CPU to GPU.
*************************************************/
GraphPath::GraphPath(Graph&_G):G(_G),StoreRoute(Task, vector<int>(1,-1)), BestRoute(Task, vector<int>())
{
	hipMalloc(&dev_edge, sizeof(Edge)*EDge);
	hipMemcpy(dev_edge, G.incL, EDge* sizeof(Edge), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_st, Task*sizeof(int));
	hipMalloc((void**)&dev_te, Task*sizeof(int));
	hipMalloc((void**)&dev_pd, Task*sizeof(float));
	hipMalloc((void**)&dev_lambda, EDge*sizeof(float));
	hipMalloc((void**)&dev_mask, Task*sizeof(int));
	hipMalloc((void**)&dev_d, Task*NODE* sizeof(float));
	hipMalloc((void**)&dev_p, Task*NODE* sizeof(int));
	hipMalloc(&dev_m, sizeof(int));
	st = new int[Task];
	te = new int[Task];
	pd = new float[Task];
	d = new float[NODE*NODE];
	pre = new int[NODE*NODE];
	lambda = new float[EDge];
	mask = new int[NODE];
	mark = new int(1);
	capacity = (float*)malloc(EDge*sizeof(float));
	for (int i = 0; i < NODE; i++)
		{
			for (int j = 0; j <NODE; j++)
			{
				if (j == i)
				{
					d[i*NODE+i] = 0.0;
					pre[i*NODE+i] = -1;
				}
				else
				{
					d[i*NODE +j] = 100000.0;
					pre[i*NODE+j] = -1;
				}
			}
		}
	hipMemcpy(dev_d, d, NODE*NODE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, pre, NODE*NODE*sizeof(int), hipMemcpyHostToDevice);
}
bool scmp(service s1,service s2)
{
	if(s1.s<s2.s)
		return true;
	return false;
}
/*************************************************
Function:bellmanFordCuda
Description:realizing LR-PROA interations on GPU.
*************************************************/
vector<pair<string,float> > GraphPath::bellmanFordCuda(vector<service>&ser,ostream& Out) {
	printf("Lagrange parrel searching..............\n");
	srand(time(NULL));
	float start = float(1000*clock())/ CLOCKS_PER_SEC;
	sort(ser.begin(),ser.end(),scmp);
	Copy2GPU(ser);
	int num = Task;
	int mum = EDge;
	int reme = 0;
	int count = 0;
	vector<RouteMark> bestroutes;
	devicesize += 2 * Task*sizeof(RouteMark);
	int bestround = 0;
	int zeor = 0;
	double totalflow = 0;
	for (int i = 0; i < Task; i++)
		totalflow += INFHOPS *pd[i];
	double bestadd = totalflow;
	float best = totalflow;
	vector<float>middata;
	for (int i = 0; i <100000000; i++)
	{
		count++;
		reme++;
		//cout<<stillS<<endl;
		dim3 blocksq(Task / threadsize + 1, NODE*Task / Task);
		ChangePameterC << <blocksq, threadsize >> >(dev_p, dev_d, dev_st);
		hipMemcpy(dev_lambda, lambda, EDge*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_mask, mask, Task*sizeof(int), hipMemcpyHostToDevice);
		dim3 blocks_square(stillS / threadsize + 1, EDge*stillS /stillS);
		//run bellman-ford algorithm on GPU.do while loop until distance array converge.
		do{
			hipMemcpy(dev_m, &zeor, sizeof(int), hipMemcpyHostToDevice);
			bellmanHigh << <blocks_square, threadsize >> >(dev_edge, dev_m, dev_d, dev_p, dev_lambda, dev_mask, stillS);
			hipMemcpy(mark, dev_m, sizeof(int), hipMemcpyDeviceToHost);
		} while (*mark);
		color << <blocks_square, threadsize >> >(dev_edge, dev_m, dev_d, dev_p, dev_lambda, dev_mask, stillS);
		hipMemcpy(pre, dev_p, sizeof(int)*NODE*NODE, hipMemcpyDeviceToHost);
		hipMemcpy(d, dev_d, sizeof(float)*NODE*NODE, hipMemcpyDeviceToHost);
		int value = rearrange2(&G, capacity, lambda, pre, d, pd, te, st,mask,bestadd, stillS, NODE, 1, StoreRoute, BestRoute);
		middata.push_back(value);
		//update the best object value
		if (value<best)
		{
			bestround = count;
			best = value;
			reme = 0;
		}
		if (stillS == 0 || reme>loomore)
			break;
	}
	float end=float(1000*clock())/ CLOCKS_PER_SEC;
	vector<pair<int, vector<int>>> result = GrabResult(BestRoute, num, mum, pd);
	int addin = result.size();
	pair<float,int> tf=CheckR(&G, result,ser,string("Lag_Parallel"));
	writejsoniter(LAGPFILE,middata,string("Lag_Parallel"));
	vector<pair<string,float>> rdata;
	rdata.push_back(make_pair(string("object"),best));
	rdata.push_back(make_pair(string("inf_obj"),totalflow));
	rdata.push_back(make_pair(string("task_add_in"),addin));
	rdata.push_back(make_pair(string("flow_add_in"),tf.first));
	rdata.push_back(make_pair(string("total_weight"),tf.second));
	rdata.push_back(make_pair(string("time"),(end-start)));
	rdata.push_back(make_pair(string("iter_num"),count));
	rdata.push_back(make_pair(string("iter_time"),float(end-start)/(float)count));
	writejsondata(DATAFILE,rdata,string("Lag_Parallel"));
	return rdata;
}
void GraphPath::CudaFree(){
	hipFree(dev_st);
	hipFree(dev_te);
	hipFree(dev_pd);
	hipFree(dev_lambda);
	hipFree(dev_mask);
	hipFree(dev_d);
	hipFree(dev_p);
	hipFree(dev_m);

}
GraphPath::~GraphPath()
{
	CudaFree();
	/*delete[] st;
	delete[] te;
	delete[] pd;
	delete[]d;
	delete[]pre;
	delete[] lambda;
	delete[] mask;
	delete mark;
	free(capacity);*/
}



